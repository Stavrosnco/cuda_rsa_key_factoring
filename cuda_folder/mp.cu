#include "hip/hip_runtime.h"
#include "mp_cuda.h"
#include <stdio.h>


#define BLOCK_WIDTH 32
#define GRID_WIDTH 10


static void HandleError( hipError_t err, const char *file, int line){
    if (err != hipSuccess) {
       printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line); 
       exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__device__ void mp_init(mp_int* res) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      res->idx[i] = 0;
   }
}

__device__ void mp_int_copy(mp_int* dest, mp_int* source) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      dest->idx[i] = source->idx[i];
   }
}

__device__ void mp_int_print_hex(mp_int* num) {

   int i;
   int print_zero = 0;

   for (i = NUM_WORDS-1; i >= 0; i--) {
      if (num->idx[i] || print_zero) {
         printf("%.8x", num->idx[i]);
         print_zero = 1;
      }
   }
}

__device__ void mp_int_gcd(mp_int* res, mp_int* lhs, mp_int* rhs) {

   int i;
   int a_even, b_even;
   int done = FALSE;
   int num_shifts = 0;
   mp_int a, b;

   mp_int_copy(&a, lhs);
   mp_int_copy(&b, rhs);

   mp_init(res);
   res->idx[NUM_WORDS-1] = 1;

   int iteration = 0;

   while (!done) {

      //printf("on iteration: %d\n", iteration++);
      //printf("last words are: %u, %u\n", a.idx[0], b.idx[0]);


      if (mp_int_is_zero(&a) || mp_int_is_zero(&b))
         break;

      a_even = mp_int_is_even(&a);
      b_even = mp_int_is_even(&b);

      if (a_even && b_even) {
         num_shifts++;
         mp_int_shift_right(&a);
         mp_int_shift_right(&b);
      }
      else if (a_even && !b_even) {
         mp_int_shift_right(&a);
      }
      else if (!a_even && b_even) {
         mp_int_shift_right(&b);
      }
      else { // both are odd

         if (mp_int_equal(&a, &b)) {
            mp_int_copy(res, &a);
            done = TRUE;
         }
         else if (mp_int_lt(&a, &b)) {
            mp_int_sub(&b, &b, &a);
            mp_int_shift_right(&b);
         }
         else {
            mp_int_sub(&a, &a, &b);
            mp_int_shift_right(&a);
         }
      }
   }

   for (i = 0; i < num_shifts; i++) {
      mp_int_shift_left(res);
   }
}

__device__ void mp_int_sub(mp_int* res, mp_int* a, mp_int* b) {

   int i, j, borrow, current; 
   mp_int lhs, rhs;

   borrow = 0;

   mp_int_copy(&lhs, a);
   mp_int_copy(&rhs, b);

   //printf("NUM_WORDS is: %d\n", NUM_WORDS);

   for (i = NUM_WORDS - 1; i >= 0; i--) {

      //printf("idx: %d, lhs: %u, rhs %u\n", i, a->idx[i], b->idx[i]);

      if (lhs.idx[i] >= rhs.idx[i]) {
         res->idx[i] = lhs.idx[i] - rhs.idx[i];
      }
      else { // need to borrow
         j = i + 1;
         //printf("start borrow idx: %d\n", j);
         while (res->idx[j] == 0) {
            res->idx[j] = UINT_MAX;
            j++;
         }
         //printf("borrowing from index %d\n", j);
         res->idx[j] -= 1;

         res->idx[i] = UINT_MAX - rhs.idx[i];
         res->idx[i] += lhs.idx[i] + 1;
      }
   }
}

__device__ void mp_int_shift_left(mp_int* res) {

   int i;

   for (i = NUM_WORDS - 1; i >= 0; i--) {

      res->idx[i] = res->idx[i] << 1;
      
      if (i > 0 && res->idx[i-1] & MOST_SIG_BIT) {
         res->idx[i] += LEAST_SIG_BIT;
      }
   }
}

__device__ void mp_int_shift_right(mp_int* res) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {

      res->idx[i] = res->idx[i] >> 1;
      
      if (i < NUM_WORDS - 1 && res->idx[i+1] & LEAST_SIG_BIT) {
         res->idx[i] += MOST_SIG_BIT;
      }
   }
}

__device__ int mp_int_gt(mp_int* lhs, mp_int* rhs) {

   int i;

   for (i = NUM_WORDS - 1; i >= 0; i--) {
      if (lhs->idx[i] > rhs->idx[i]) {
         return TRUE;
      }
      else if (rhs->idx[i] > lhs->idx[i]) {
         return FALSE;
      }
   }

   return FALSE;
}

__device__ int mp_int_gte(mp_int* lhs, mp_int* rhs) {

   return (!mp_int_gt(rhs, lhs));
}

__device__ int mp_int_lt(mp_int* lhs, mp_int* rhs) {

   return mp_int_gt(rhs, lhs);
}

__device__ int mp_int_lte(mp_int* lhs, mp_int* rhs) {

   return (!mp_int_gt(lhs, rhs));
}

__device__ int mp_int_is_odd(mp_int* num) {

   return (num->idx[0] & 1);
}

__device__ int mp_int_is_even(mp_int* num) {

   return (!mp_int_is_odd(num));
}

__device__ int mp_int_is_zero(mp_int* num) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      if (num->idx[i]) {
         return FALSE;
      }
   }

   return TRUE;
}

__device__ int mp_int_equal(mp_int* a, mp_int* b) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      if (a->idx[i] != b->idx[i])
         return FALSE;
   }

   return TRUE;
}

__global__ void mp_kernel(mp_int* res, mp_int* keys, int num_keys, int res_width, int idx_x, int idx_y) {
   
   int row = blockIdx.y * blockDim.y + threadIdx.y + idx_x;
   int col = blockIdx.x * blockDim.x + threadIdx.x + idx_y;
   int res_row = blockIdx.y * blockDim.y + threadIdx.y; 
   int res_col = blockIdx.x * blockDim.x + threadIdx.x; 
   

   if(row > col && row < num_keys){ 
      mp_int_gcd(&res[res_row*res_width + res_col], &keys[row], &keys[col]);
      //check that it equals 1   
   
   }
   __syncthreads();
   


}


void cuda_call(int num_keys, mp_int *keys, mp_int *res){


   int idx_x, idx_y, i,j;
   int num_calls;
   
   

   mp_int *keys_d; 
   mp_int *res_d;
   num_calls = num_keys/(BLOCK_WIDTH*GRID_WIDTH);
   if(num_keys % (BLOCK_WIDTH * GRID_WIDTH)){
      num_calls++;
   }

   dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH);
   dim3 dimGrid(GRID_WIDTH,GRID_WIDTH);
   HANDLE_ERROR(hipMalloc((void **) &keys_d, sizeof(mp_int)*num_keys));
   HANDLE_ERROR(hipMemcpy(keys_d, keys, sizeof(mp_int)*num_keys, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc((void **) &res_d, sizeof(mp_int)*(GRID_WIDTH*GRID_WIDTH*BLOCK_WIDTH*BLOCK_WIDTH)));

  printf("in cuda call num calls %d\n", num_calls);  
   for(i=0; i<num_calls; i+=(BLOCK_WIDTH*GRID_WIDTH)) {
      
      for(j=0; j<num_calls; j+= (BLOCK_WIDTH *GRID_WIDTH)) {
       
         if(i >= j) {
      
            mp_kernel<<<dimGrid,dimBlock>>>(res_d, keys_d, num_keys, BLOCK_WIDTH * GRID_WIDTH, i, j);
            
            HANDLE_ERROR(hipMemcpy(res, res_d, sizeof(mp_int)*num_keys, hipMemcpyDeviceToHost));
            printf("returned\n"); 
         }
      }
   }

}
int main(void){
  


   printf("hello world\n"); 
   cuda_call(10, NULL, NULL);  
   return 0; 

}


