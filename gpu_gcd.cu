#include "hip/hip_runtime.h"
#include "mp_cuda.h"

int parse_largeint_file(char* filename, mp_int* intlist, int max_size, int verbose) {

   FILE* file = fopen(filename, "r");

   mpz_t tmp;
   mpz_init(tmp);

   int i = 0;

   while (mpz_inp_str(tmp, file, 10) > 0 && i < max_size) {

      if (verbose) {
         printf("n%d = ", i+1);
         mpz_out_str(stdout, 10, tmp);
         printf("\n\n");
      }

      mp_init(&intlist[i]);
      mp_import_mpz(&intlist[i], tmp);

      i++;
   }

   fclose(file);

   return i;
}

void print_mpz_hex(char* description, mpz_t num) {

   printf("%s", description);
   mpz_out_str(stdout, 16, num);
   printf("\n");
}

void print_mp_hex(char* description, mp_int* num) {

   printf("%s", description);
   mp_int_print_hex(num);
   printf("\n");
}

void mp_import_mpz(mp_int* dest, mpz_t source) {

   size_t count;

   mpz_export(&dest->idx[0], &count, -1, sizeof(unsigned int), 0, 0, source);
}

void mp_export_mpz(mpz_t dest, mp_int* source) {

   mpz_import(dest, NUM_WORDS, -1, sizeof(unsigned int), 0, 0, &source->idx[0]);
}



__host__ __device__ void mp_init(mp_int* res) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      res->idx[i] = 0;
   }
}

__device__ void mp_int_copy(mp_int* dest, mp_int* source) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      dest->idx[i] = source->idx[i];
   }
}

__host__ __device__ void mp_int_print_hex(mp_int* num) {

   int i;
   int print_zero = 0;

   for (i = NUM_WORDS-1; i >= 0; i--) {
      if (num->idx[i] || print_zero) {
         printf("%.8x", num->idx[i]);
         print_zero = 1;
      }
   }
}

__device__ void mp_int_gcd(mp_int* res, mp_int* lhs, mp_int* rhs) {

   int i;
   int a_even, b_even;
   int done = FALSE;
   int num_shifts = 0;
   mp_int a, b;

   mp_int_copy(&a, lhs);
   mp_int_copy(&b, rhs);

   mp_init(res);
   res->idx[NUM_WORDS-1] = 1;

  

   while (!done) {

      //printf("on iteration: %d\n", iteration++);
      //printf("last words are: %u, %u\n", a.idx[0], b.idx[0]);


      if (mp_int_is_zero(&a) || mp_int_is_zero(&b))
         break;

      a_even = mp_int_is_even(&a);
      b_even = mp_int_is_even(&b);

      if (a_even && b_even) {
         num_shifts++;
         mp_int_shift_right(&a);
         mp_int_shift_right(&b);
      }
      else if (a_even && !b_even) {
         mp_int_shift_right(&a);
      }
      else if (!a_even && b_even) {
         mp_int_shift_right(&b);
      }
      else { // both are odd

         if (mp_int_equal(&a, &b)) {
            mp_int_copy(res, &a);
            done = TRUE;
         }
         else if (mp_int_lt(&a, &b)) {
            mp_int_sub(&b, &b, &a);
            mp_int_shift_right(&b);
         }
         else {
            mp_int_sub(&a, &a, &b);
            mp_int_shift_right(&a);
         }
      }
   }

   for (i = 0; i < num_shifts; i++) {
      mp_int_shift_left(res);
   }
}

__device__ void mp_int_sub(mp_int* res, mp_int* a, mp_int* b) {

   int i, j; 
   mp_int lhs, rhs;


   mp_int_copy(&lhs, a);
   mp_int_copy(&rhs, b);

   //printf("NUM_WORDS is: %d\n", NUM_WORDS);

   for (i = NUM_WORDS - 1; i >= 0; i--) {

      //printf("idx: %d, lhs: %u, rhs %u\n", i, a->idx[i], b->idx[i]);

      if (lhs.idx[i] >= rhs.idx[i]) {
         res->idx[i] = lhs.idx[i] - rhs.idx[i];
      }
      else { // need to borrow
         j = i + 1;
         //printf("start borrow idx: %d\n", j);
         while (res->idx[j] == 0) {
            res->idx[j] = UINT_MAX;
            j++;
         }
         //printf("borrowing from index %d\n", j);
         res->idx[j] -= 1;

         res->idx[i] = UINT_MAX - rhs.idx[i];
         res->idx[i] += lhs.idx[i] + 1;
      }
   }
}

__device__ void mp_int_shift_left(mp_int* res) {

   int i;

   for (i = NUM_WORDS - 1; i >= 0; i--) {

      res->idx[i] = res->idx[i] << 1;
      
      if (i > 0 && res->idx[i-1] & MOST_SIG_BIT) {
         res->idx[i] += LEAST_SIG_BIT;
      }
   }
}

__device__ void mp_int_shift_right(mp_int* res) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {

      res->idx[i] = res->idx[i] >> 1;
      
      if (i < NUM_WORDS - 1 && res->idx[i+1] & LEAST_SIG_BIT) {
         res->idx[i] += MOST_SIG_BIT;
      }
   }
}

__device__ int mp_int_gt(mp_int* lhs, mp_int* rhs) {

   int i;

   for (i = NUM_WORDS - 1; i >= 0; i--) {
      if (lhs->idx[i] > rhs->idx[i]) {
         return TRUE;
      }
      else if (rhs->idx[i] > lhs->idx[i]) {
         return FALSE;
      }
   }

   return FALSE;
}

__device__ int mp_int_gte(mp_int* lhs, mp_int* rhs) {

   return (!mp_int_gt(rhs, lhs));
}

__device__ int mp_int_lt(mp_int* lhs, mp_int* rhs) {

   return mp_int_gt(rhs, lhs);
}

__device__ int mp_int_lte(mp_int* lhs, mp_int* rhs) {

   return (!mp_int_gt(lhs, rhs));
}

__device__ int mp_int_is_odd(mp_int* num) {

   return (num->idx[0] & 1);
}

__device__ int mp_int_is_even(mp_int* num) {

   return (!mp_int_is_odd(num));
}

__device__ int mp_int_is_zero(mp_int* num) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      if (num->idx[i]) {
         return FALSE;
      }
   }

   return TRUE;
}

__device__ int mp_int_equal(mp_int* a, mp_int* b) {

   int i;

   for (i = 0; i < NUM_WORDS; i++) {
      if (a->idx[i] != b->idx[i])
         return FALSE;
   }

   return TRUE;
}

__global__ void mp_kernel(result_keys *res, mp_int* keys, int num_keys, int res_width, int idx_x, int idx_y) {
   
   
   int tid_y = blockIdx.y * blockDim.y + threadIdx.y; 
   int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
   int row = tid_y + idx_y;
   int col = tid_x + idx_x;  
   mp_int cf; 
   mp_init(&cf); 

   mp_int one; 
   mp_init(&one);
   one.idx[0] = 1; 
    

   

   if(tid_x > tid_y && row < num_keys && col < num_keys){ 
      mp_int_gcd(&cf, &keys[col], &keys[row]);
      if(mp_int_gt(&cf, &one)){
         res[tid_y * res_width + tid_x].idx_a = col; 
         res[tid_y * res_width + tid_x].idx_b = row;   
	      
      }
      else{
         res[tid_y * res_width + tid_x].idx_a = 0; 
         res[tid_y * res_width + tid_x].idx_b = 0;  
	   }
   }
   else{
         res[tid_y * res_width + tid_x].idx_a = 0; 
         res[tid_y * res_width + tid_x].idx_b = 0;  
	}
   __syncthreads();
   


}


void cuda_call(int num_keys, mp_int *keys, result_keys *res){


   int i,j, k, idx_x,idx_y;
   int num_calls;
   
   

   mp_int *keys_d; 
   result_keys  *res_d;
   
   mpz_t tmp;
   mpz_init(tmp);
   
   
   
   
   num_calls = num_keys/RES_WIDTH;
   if(num_keys % RES_WIDTH){
      num_calls++;
   }
   
            

   dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH);
   dim3 dimGrid(GRID_WIDTH,GRID_WIDTH);
   HANDLE_ERROR(hipMalloc((void **) &keys_d, sizeof(mp_int)*num_keys));
   HANDLE_ERROR(hipMemcpy(keys_d, keys, sizeof(mp_int)*num_keys, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc((void **) &res_d, sizeof(result_keys)*RES_SIZE));

  //printf("in cuda call num calls %d\n", num_calls);  
  //printf("result width = %d\n", RES_WIDTH);
  idx_x = 0;  
  //printf("result size = %d num_keys %d \n", RES_SIZE, num_keys);
   for(i=0; i<num_calls; i++) {
      idx_y =0; 
      
      for(j=0; j<num_calls; j++) {
       
         if(idx_x <= idx_y) {
         
      	  // printf("idx_x %d idx_y %d\n", idx_x, idx_y); 
            mp_kernel<<<dimGrid,dimBlock>>>(res_d, keys_d, num_keys, RES_WIDTH, idx_x, idx_y);
            
            HANDLE_ERROR(hipMemcpy(res, res_d, sizeof(result_keys)*RES_SIZE, hipMemcpyDeviceToHost));
           
            
            for(k=0; k< RES_SIZE; k++){
                if(res[k].idx_a != 0 || res[k].idx_b != 0){
            	   //printf(" at k = %d a %u, b %u \n", k, res[k].idx_a, res[k].idx_b);ß 
            	   mp_export_mpz(tmp, &keys[res[k].idx_b]);
                  mpz_out_str(stdout, 10, tmp);
                  printf("\n"); 
                  mp_export_mpz(tmp, &keys[res[k].idx_a]);
                  mpz_out_str(stdout, 10, tmp);
                  printf("\n"); 
            	 }
            } 
           // printf("returned\n"); 
            
         }
         
         idx_y += RES_WIDTH;
      }
      idx_x += RES_WIDTH;
      
   }
   

}
  



int main(int argc, char** argv) {

   
   result_keys *comp_key_idxs;
   int size=0;
   char filename[FILE_MAX];
   mp_int *intlist = (mp_int *) calloc(sizeof(mp_int)*MAX_SIZE, 1);

   verify_arguments(argc, argv, filename);

   size = parse_largeint_file(filename, intlist, MAX_SIZE, 0);
   
   comp_key_idxs = (result_keys*) calloc(RES_SIZE * sizeof(result_keys), 1); 
  // compromizable_pairs = pairwise_gcd(intlist, size, &comp_key_idxs, 1);
   cuda_call(size, intlist, comp_key_idxs); 
   free(intlist);

   return 0;
}

void verify_arguments(int argc, char** argv, char* filename) {

   if (argc < 2) {
      fprintf(stderr, "invalid argument list, requires: %s filename\n", argv[0]);
      exit(0);
   }
   else {
      strncpy(filename, argv[1], FILE_MAX - 1);
   }
}

